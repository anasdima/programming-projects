#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include <time.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>

#define BlockSize 512

void random_initialization(knn_struct *set, int cal){

 int i = 0;
 int n = set->leading_dim;
 int m = set->secondary_dim;
 float *tmp_set = set->data;

 srand(cal*time(NULL));
 /*Generate random floating points [-50 50]*/
 for(i=0; i<m*n; i++){
 
   tmp_set[i] = 100 * (float)rand() / RAND_MAX - 50; 

 }

}

void save_d(float* data, char* file, int N, int M){

  FILE *outfile;
  
  printf("Saving data to file: %s\n", file);

  if((outfile=fopen(file, "wb")) == NULL){
    printf("Can't open output file");
  }

  fwrite(data, sizeof(float), N*M, outfile);

  fclose(outfile);

}

void save_int(int* data, char* file, int N, int M){

  FILE *outfile;
  
  printf("Saving data to file: %s\n", file);

  if((outfile=fopen(file, "wb")) == NULL){
    printf("Can't open output file");
  }

  fwrite(data, sizeof(int), N*M, outfile);

  fclose(outfile);

}

void clean(knn_struct* d){

  free(d->data);
}

__device__ float euclidean_distance(float *X, float *Y, int N){

	int i = 0;
	float dst = 0;

#pragma unroll 2
	for(i=0; i<N; i++){
		float tmp = (X[i] - Y[i]);
		dst += tmp * tmp;
	}

  return(dst);
}

__global__ void compute_distance(float* query, float* data, float* dist, int* idx, int numObjects, int numAttributes){

	extern __shared__ float Qs[];
	
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int element = tid;
		
	__syncthreads();
		
	if(tid<numObjects) {
		if(threadIdx.x<numAttributes) {  /* load query in shared memory */
			Qs[threadIdx.x] = query[threadIdx.x];
		}
			
		__syncthreads();

		dist[element] = euclidean_distance(data + element*numAttributes, Qs, numAttributes);	
		idx[element] = tid;
	}
}

__global__ void reduce(float* dist, int* idx, int N, int stride){

	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int element = tid;
		
	if(tid < (stride) ) {
	
		if(dist[element]>dist[element+stride]) {
		
			dist[element] = dist[element+stride];
			idx[element] = idx[element+stride];
		}
	}

}

void knns(knn_struct* queries, knn_struct* dataset, float *NNdist, int *NNidx, int k){

	float *d_dist,*d_tmp_dist;
	int *d_idx,*d_tmp_idx;
	int numQueries = queries->secondary_dim;
	int numObjects = dataset->secondary_dim;
	int numAttributes = dataset->leading_dim;
	int i,j;
	unsigned long stride=0;
	float max = FLT_MAX;
		
	hipMalloc((void**)&d_dist, numObjects*sizeof(float));
	hipMalloc((void**)&d_idx, numObjects*sizeof(int));
	
	// Used for parallel reduction	
	hipMalloc((void**)&d_tmp_dist, numObjects*sizeof(float));
	hipMalloc((void**)&d_tmp_idx, numObjects*sizeof(int));
	float h_tmp_dist[2];
	int h_tmp_idx[2];
  
	float tmp_grid_size = (int) ceil((float)numObjects/(float)BlockSize);
	float tmp_block_size = numObjects<BlockSize ? numObjects:BlockSize;

	dim3 grid((int)tmp_grid_size,1);
	dim3 threads((int)tmp_block_size, 1);  
	
	printf("Grid size: %f | Block size: %f\n",tmp_grid_size,tmp_block_size);
	
	for(i=0;i<numQueries;i++) {
		
		compute_distance<<< grid, threads, numAttributes*sizeof(float) >>>(queries->data+i*numAttributes, dataset->data, d_dist, d_idx, numObjects, numAttributes);	
		
		float distance;
		hipMemcpy(&distance, d_dist, 1*sizeof(float), hipMemcpyDeviceToHost);
	
		/* Find k nearest neighbours */
		for(j=0;j<k;j++) {
		
			/* Set reduce grid parameters and stride */
			tmp_grid_size = ceil((float)numObjects/((float)BlockSize*2.0));	
			dim3 reduce_grid((int)tmp_grid_size);
			if(numObjects%2 == 0) {
				stride = numObjects/2;
			}
			else {
				stride = numObjects/2 + 1;				
			}
	
			/* Copy distances and indexes to temp, editable memory */
			hipMemcpy(d_tmp_dist, d_dist, numObjects*sizeof(float), hipMemcpyDeviceToDevice);
			hipMemcpy(d_tmp_idx, d_idx, numObjects*sizeof(int), hipMemcpyDeviceToDevice);

			/* Find minimum distance using parallel reduction */
			while(stride > 1) {
						
				reduce<<< reduce_grid, threads >>>(d_tmp_dist, d_tmp_idx, numObjects, stride); // Global sync point
				
				tmp_grid_size = ceil((float)tmp_grid_size/2.0);
				dim3 reduce_grid((int)tmp_grid_size);
				
				if(stride%2 == 0) {
					stride = stride/2;
				}
				else {
					stride = stride/2 + 1;				
				}
				
			}
					
			hipMemcpy(h_tmp_dist, d_tmp_dist, 2*sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(h_tmp_idx, d_tmp_idx, 2*sizeof(int), hipMemcpyDeviceToHost);
						
			if(h_tmp_dist[0] < h_tmp_dist[1]) {
			
				NNdist[i*k+j] = h_tmp_dist[0];
				NNidx[i*k+j] = h_tmp_idx[0];
				
			}
			else {
			
				NNdist[i*k+j] = h_tmp_dist[1];
				NNidx[i*k+j] = h_tmp_idx[1];
				
			}
					
			hipMemcpy(d_dist+NNidx[i*k+j], &max, sizeof(float), hipMemcpyHostToDevice); // Exclude this minimum distance for the next minimum search
			
		}
		
	}
	
	hipFree(d_dist);
	hipFree(d_tmp_dist);
	hipFree(d_idx);
	hipFree(d_tmp_idx);
}

int main(int argc, char **argv){

	int numObjects = atoi(argv[1]);
	int numDim = atoi(argv[2]);
	int numQueries = atoi(argv[3]);
	int k = atoi(argv[4]);

	printf("objects: %d\n", numObjects);
	printf("dimentions: %d\n", numDim);
	printf("queries: %d\n", numQueries);
	printf("k: %d\n", k);

	/*===== Host ======*/
	struct timeval first, second, lapsed;
	struct timezone tzp;
  
	//size_t memory_free, memory_total;

	char *dataset_file = "training_set.bin";
	char *query_file = "query_set.bin";
	char *KNNdist_file = "KNNdist.bin";
	char *KNNidx_file = "KNNidx.bin" ;

	knn_struct training_set;
	knn_struct query_set;
	float *NNdist;
	int *NNidx;
  
	/*==== Device ======*/
	knn_struct d_training_set;
	knn_struct d_query_set;
  
	/*======== Initialization =======*/
	training_set.leading_dim = numDim;
	training_set.secondary_dim = numObjects;
	query_set.leading_dim = numDim;
	query_set.secondary_dim = numQueries;
  
	d_training_set.leading_dim = numDim;
	d_training_set.secondary_dim = numObjects;
	d_query_set.leading_dim = numDim;
	d_query_set.secondary_dim = numQueries;
  
	/*======== Host memory allocation ======*/
	training_set.data = (float*)malloc(numObjects*numDim*sizeof(float));
	query_set.data = (float*)malloc(numQueries*numDim*sizeof(float));
	NNdist = (float*)malloc(numQueries*k*sizeof(float));
	NNidx = (int*)malloc(numQueries*k*sizeof(int));  
 
	/*========= Device memory allocation======*/
	hipMalloc((void **)&d_training_set.data, training_set.leading_dim*training_set.secondary_dim*sizeof(float));
	hipMalloc((void**)&d_query_set.data, query_set.leading_dim*query_set.secondary_dim*sizeof(float));
  
	/*======== Initialize =========*/
	random_initialization(&training_set, 1);
	random_initialization(&query_set, 2);
  
	/*========= Device memory initialization =========*/
	hipMemcpy(d_training_set.data, training_set.data, training_set.leading_dim*training_set.secondary_dim*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_query_set.data, query_set.data, query_set.leading_dim*query_set.secondary_dim*sizeof(float), hipMemcpyHostToDevice);
  
	gettimeofday(&first, &tzp);

	knns(&d_query_set, &d_training_set, NNdist, NNidx, k);

	gettimeofday(&second, &tzp);
  
	if(first.tv_usec>second.tv_usec){
		second.tv_usec += 1000000;
		second.tv_sec--;
	}
  
	lapsed.tv_usec = second.tv_usec - first.tv_usec;
	lapsed.tv_sec = second.tv_sec - first.tv_sec;

	printf("Time elapsed: %d, %d s\n", lapsed.tv_sec, lapsed.tv_usec); 

	save_d(query_set.data, query_file, numQueries, numDim);
	save_d(training_set.data, dataset_file, numObjects, numDim);
	save_d(NNdist, KNNdist_file, k, numQueries);
	save_int(NNidx, KNNidx_file, k, numQueries);

	/*===== clean memory ========*/
	clean(&training_set);
	clean(&query_set);
	free(NNdist);
	free(NNidx);

}




