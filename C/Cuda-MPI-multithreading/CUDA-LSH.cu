#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#define HASH_STEP 720
#define WARP_SIZE 32

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void generate_randoms(float *d_random_r, int numDim, unsigned long seed) {

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
    hiprandState localState;
    hiprand_init (seed, idx, 0, &localState);
    
    for(int i=0;i<numDim;i++) {

    	d_random_r[idx*(numDim+1)+i] = hiprand_uniform(&localState); // ri [0,1]
    }

    d_random_r[idx*(numDim+1)+numDim] = hiprand_uniform(&localState)*HASH_STEP; // bi [0,720]
}

__device__ float matrix_multiplication(float *d_data, float *d_random_r, int numDim) {

	float sum = 0;

	for(int i=0;i<numDim;i++) {

		sum += d_data[i]*d_random_r[i];
	}

	return(sum);

}

__global__ void hash_f(float *d_data, int *d_hash_codes, float *d_random_r, int numDim, unsigned long seed) {

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    d_hash_codes[idx] = floorf((matrix_multiplication(&d_data[(idx/numDim)*numDim],&d_random_r[idx*(numDim+1)],numDim)
    				+ d_random_r[idx*(numDim+1)+numDim])/HASH_STEP);
}

__global__ void initial_count(int *d_hash_codes, int *d_offsets, int digit, int numDim, int NTHREADS) {

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int s_zeros = 0,s_ones = 0;

	for(int i=0;i<64;i++) {

		if(d_hash_codes[idx*numDim*64+i*numDim+numDim-digit] == 0) {
			s_zeros++;
		}
		else if(d_hash_codes[idx*numDim*64+i*numDim+numDim-digit] == 1){
			s_ones++;
		}
	}

	d_offsets[idx*2+2] = s_zeros; // First 2 slots are 0 in the table
	d_offsets[idx*2+3] = s_ones;  // Same

}

__global__ void lsd_count(int *d_hash_codes, int *d_offsets, int *d_ids, int digit, int numDim, int NTHREADS) {

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int s_zeros = 0,s_ones = 0;

	for(int i=0;i<64;i++) {

		if(d_hash_codes[d_ids[idx*64+i]*numDim+numDim-digit] == 0) {
			s_zeros++;
		}
		else if(d_hash_codes[d_ids[idx*64+i]*numDim+numDim-digit] == 1){
			s_ones++;
		}
	}

	d_offsets[idx*2+2] = s_zeros; // First 2 slots are 0 in the table
	d_offsets[idx*2+3] = s_ones;  // Same

}

__global__ void initialize_sorting(int *d_hash_codes, int *d_offsets, int *d_ids_1, int digit, int numDim, int NTHREADS) {

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int offset_zero = d_offsets[idx*2];
	int offset_one = d_offsets[idx*2+1]+d_offsets[2*NTHREADS]; // d_offsets[2*NTHREADS] holds the total number of zeros

	for(int i=0;i<64;i++) {

		if(d_hash_codes[idx*numDim*64+i*numDim+numDim-digit] == 0) {

			d_ids_1[offset_zero] = idx*64+i;
			offset_zero++;

		}
		else if(d_hash_codes[idx*numDim*64+i*numDim+numDim-digit] == 1){

			d_ids_1[offset_one] = idx*64+i;
			offset_one++;

		}

	}

	// At this point offset_zero + offset_one = 64

}

__global__ void radix_sort(int *d_hash_codes, int *d_offsets, int *d_ids_1, int *d_ids_2, int digit, int numDim, int NTHREADS) {

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int offset_zero = d_offsets[idx*2];
	int offset_one = d_offsets[idx*2+1]+d_offsets[2*NTHREADS]; // d_offsets[2*NTHREADS] holds the total number of zeros

	for(int i=0;i<64;i++) {

		if(d_hash_codes[d_ids_1[idx*64+i]*numDim+numDim-digit] == 0) {

			d_ids_2[offset_zero] = d_ids_1[idx*64+i];
			offset_zero++;

		}
		else if(d_hash_codes[d_ids_1[idx*64+i]*numDim+numDim-digit] == 1){

			d_ids_2[offset_one] = d_ids_1[idx*64+i];
			offset_one++;

		}

	}

	// At this point offset_zero + offset_one = 64

}

__global__ void write_sorted_hash_codes(int *d_hash_codes, int *d_sorted_hash_codes, int *d_ids, int numDim) {

	int idx = threadIdx.x + blockIdx.x*blockDim.x;

	for(int i=0;i<64;i++) {
		for(int j=0;j<numDim;j++) {
			d_sorted_hash_codes[idx*64*numDim+i*numDim+j] = d_hash_codes[d_ids[idx*64+i]*numDim+j];
		}
	}
}

void lsh(int numObjects, int numDim, float *h_data) {

	/* Generate hash codes */

	int NTHREADS = ceil((float)numObjects*numDim/(float)(64*WARP_SIZE)); //Concurrent threads per loop
	int BLOCK_SIZE = 256;
	int GRID_SIZE = ceil((float)NTHREADS/(float)BLOCK_SIZE);

	dim3 grid(GRID_SIZE,1);
	dim3 threads(BLOCK_SIZE,1);

	printf("NTHREADS:%d GRID_SIZE:%d BLOCK_SIZE:%d\n", NTHREADS,GRID_SIZE,BLOCK_SIZE);

	float *d_random_r;
	hipMalloc((void**)&d_random_r,NTHREADS*(numDim+1)*sizeof(float)); // numDim+1 because ri has numDim values and bi has 1

	float *d_data;
	hipMalloc((void**)&d_data,numObjects*numDim*sizeof(float));
	hipMemcpy(d_data,h_data,numObjects*numDim*sizeof(float),hipMemcpyHostToDevice);

	int *d_hash_codes;
	hipMalloc((void**)&d_hash_codes, numObjects*numDim*sizeof(int));

    for(int i=0;i<64*WARP_SIZE;i++) {

    	generate_randoms<<<grid,threads>>>(d_random_r,numDim,time(NULL));

    	hash_f<<<grid,threads>>>(&d_data[i*NTHREADS],&d_hash_codes[i*NTHREADS],d_random_r,numDim,time(NULL));

    }

    hipFree(d_random_r);
    hipFree(d_data);

    int *h_hash_codes;
	h_hash_codes = (int *)malloc(numObjects*numDim*sizeof(int));

	hipMemcpy(h_hash_codes,d_hash_codes,numObjects*numDim*sizeof(int),hipMemcpyDeviceToHost);

    /* Sort hash codes */

    NTHREADS = ceil((float)numObjects/(float)64); //Concurrent threads per loop
	BLOCK_SIZE = 256;
	GRID_SIZE = ceil((float)NTHREADS/(float)BLOCK_SIZE);

	dim3 grid_2(GRID_SIZE,1);
	dim3 threads_2(BLOCK_SIZE,1);

	printf("NTHREADS:%d GRID_SIZE:%d BLOCK_SIZE:%d\n", NTHREADS,GRID_SIZE,BLOCK_SIZE);

	int *d_ids_1;
	hipMalloc((void**)&d_ids_1,numObjects*sizeof(int));

	int *d_ids_2;
	hipMalloc((void**)&d_ids_2,numObjects*sizeof(int));

	int *d_offsets;
	hipMalloc((void**)&d_offsets,(2*NTHREADS+2)*sizeof(int));

	int *d_sorted_hash_codes;
	hipMalloc((void**)&d_sorted_hash_codes, numObjects*numDim*sizeof(int));

	initial_count<<<grid_2,threads_2>>>(d_hash_codes,d_offsets,1,numDim,NTHREADS);

	hipDeviceSynchronize();

	int *h_offsets;
	h_offsets = (int *)malloc((2*NTHREADS+2)*sizeof(int));

	hipMemcpy(h_offsets,d_offsets,(2*NTHREADS+2)*sizeof(int),hipMemcpyDeviceToHost);

	//Calculate offsets serially

	h_offsets[0] = 0;
	h_offsets[1] = 0;

	for(int j=2;j<(2*NTHREADS+2);j++) {

			h_offsets[j] += h_offsets[j-2];
	}

	hipMemcpy(d_offsets,h_offsets,(2*NTHREADS+2)*sizeof(int),hipMemcpyHostToDevice);

	initialize_sorting<<<grid_2,threads_2>>>(d_hash_codes,d_offsets,d_ids_1,1,numDim,NTHREADS);

	for(int i=1;i<numDim;i++) {

		if (i%2!=0) {

			lsd_count<<<grid_2,threads_2>>>(d_hash_codes,d_offsets,d_ids_1,(i+1),numDim,NTHREADS);

			hipMemcpy(h_offsets,d_offsets,(2*NTHREADS+2)*sizeof(int),hipMemcpyDeviceToHost);

			//Calculate offsets serially
			h_offsets[0] = 0;
			h_offsets[1] = 0;

			for(int j=2;j<(2*NTHREADS+2);j++) {

				h_offsets[j] += h_offsets[j-2];
			}

			hipMemcpy(d_offsets,h_offsets,(2*NTHREADS+2)*sizeof(int),hipMemcpyHostToDevice);

			radix_sort<<<grid_2, threads_2>>>(d_hash_codes,d_offsets,d_ids_1,d_ids_2,(i+1),numDim,NTHREADS);

		}
		else {

			lsd_count<<<grid_2,threads_2>>>(d_hash_codes,d_offsets,d_ids_2,(i+1),numDim,NTHREADS);

			hipMemcpy(h_offsets,d_offsets,(2*NTHREADS+2)*sizeof(int),hipMemcpyDeviceToHost);

			//Calculate offsets serially

			h_offsets[0] = 0;
			h_offsets[1] = 0;

			for(int j=2;j<(2*NTHREADS+2);j++) {

					h_offsets[j] += h_offsets[j-2];
			}

			hipMemcpy(d_offsets,h_offsets,(2*NTHREADS+2)*sizeof(int),hipMemcpyHostToDevice);

			radix_sort<<<grid_2, threads_2>>>(d_hash_codes,d_offsets,d_ids_2,d_ids_1,(i+1),numDim,NTHREADS);
				
		}

	}

	write_sorted_hash_codes<<<grid_2, threads_2>>>(d_hash_codes,d_sorted_hash_codes,d_ids_2,numDim);

	int *h_sorted_hash_codes;
	h_sorted_hash_codes = (int *)malloc(numObjects*numDim*sizeof(int));

	hipMemcpy(h_sorted_hash_codes,d_sorted_hash_codes,numObjects*numDim*sizeof(int),hipMemcpyDeviceToHost);

}

int main(int argc, char** argv) {

	int numObjects = atoi(argv[1]);
	int numDim = atoi(argv[2]);

	FILE *dataset;
	float *h_data;
	h_data = (float *)malloc(numObjects*numDim*sizeof(float));

	dataset = fopen("/export/home/dhmtasos/Ergasia4/data.bin", "rb");
	if(dataset == NULL) {
		printf("Error opening data.bin\n");
	}

	size_t a = fread(h_data, sizeof(float), numObjects*numDim, dataset);
	if(a!=numObjects*numDim) {
		printf("Error reading data from data.bin\n");
	}

	fclose(dataset);

	struct timeval first, second, lapsed;
	struct timezone tzp;

	gettimeofday(&first, &tzp);

	lsh(numObjects,numDim,h_data);

	gettimeofday(&second, &tzp);

	if(first.tv_usec>second.tv_usec){
		second.tv_usec += 1000000;
		second.tv_sec--;
	}
  
	lapsed.tv_usec = second.tv_usec - first.tv_usec;
	lapsed.tv_sec = second.tv_sec - first.tv_sec;

	printf("Time elapsed: %d, %d s\n", lapsed.tv_sec, lapsed.tv_usec);

}